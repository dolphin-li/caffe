#include "hip/hip_runtime.h"
#pragma once

#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

//#define IMAGE_DEBUG

#ifdef IMAGE_DEBUG
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#endif

namespace caffe {

	template <typename Dtype>
	__global__ void rgb2luma_forward(Dtype* dst, const Dtype*src,
		const int nsample,
		const int npixel, const int nchannel, 
		const Dtype Mb, const Dtype Mg, const Dtype Mr, const Dtype T) 
	{
		CUDA_KERNEL_LOOP(index, npixel*nsample) {
			const int isample = index/npixel;
			const int ipixel = index%npixel;
			const int srcIndex = isample*npixel*nchannel + ipixel;
			dst[index] = Mb * src[srcIndex] + Mg * src[srcIndex+npixel] + Mr * src[srcIndex+npixel+npixel] + T;
		}
	}

	template <typename Dtype>
	Dtype Bgr2LumaLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		vector<Blob<Dtype>*>* top) 
	{
		const Dtype scale = layer_param_.bgr2luma_param().scale();
		// bgr
		const Dtype M[3] = {24.966f/255.f, 128.553f/255.f, 65.481f/255.f};
		const Dtype T = 16.f * scale;

		for(size_t i = 0; i<bottom.size(); i++)
		{
			const Dtype* bottom_data = bottom[i]->gpu_data();
			Dtype* top_data = (*top)[i]->mutable_gpu_data();
			const int n = bottom[i]->num();
			const int c = bottom[i]->channels();
			const int h = bottom[i]->height();
			const int w = bottom[i]->width();
			const int cl = h*w;

			rgb2luma_forward<Dtype><<<CAFFE_GET_BLOCKS(cl*n), CAFFE_CUDA_NUM_THREADS>>>(
				top_data, bottom_data, n, cl, c, M[0], M[1], M[2], T);
			CUDA_POST_KERNEL_CHECK;

#ifdef IMAGE_DEBUG
			cv::Mat img;
			img.create(h, w, CV_8UC1);
			for(int ni=0; ni<n; ni++)
			{
				const Dtype* pSrc = (*top)[i]->cpu_data() + ni*h*w;
				for(int p=0; p<cl; p++)
				{
					img.at<char>(p/w, p%w) = pSrc[p]*255;
				}
				cv::imshow("luma", img);
				cvWaitKey();
			}
#endif

		}
		return Dtype(0.);
	}

	INSTANTIATE_CLASS(Bgr2LumaLayer);

}  // namespace caffe
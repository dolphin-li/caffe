#include "hip/hip_runtime.h"
#pragma once

#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

//#define IMAGE_DEBUG
#ifdef IMAGE_DEBUG
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#endif

namespace caffe {

	template <typename Dtype>
	Dtype AddNoiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		vector<Blob<Dtype>*>* top) 
	{
		for(size_t i = 0; i<bottom.size(); i++)
		{
			// generate noise-levels
			generate_noise_levels();
			const Dtype* noise_level_data = noise_levels_.gpu_data();
			const Dtype* bottom_data = bottom[i]->gpu_data();
			Dtype* top_data = (*top)[i*2]->mutable_gpu_data();
			Dtype* top_nl = (*top)[i*2+1]->mutable_gpu_data();
			Dtype* noise_data = noises_.mutable_gpu_data();

			const int n = bottom[i]->num();
			const int np = bottom[i]->channels()*bottom[i]->height()*bottom[i]->width();
			caffe_gpu_rng_gaussian(n*np, Dtype(0), Dtype(1), noise_data);
			caffe_gpu_dgmm(HIPBLAS_SIDE_RIGHT, np, n, noise_data, noise_level_data, noise_data);
			caffe_gpu_add(n*np, bottom_data, noise_data, top_data);

			caffe_gpu_copy(noise_levels_.count(), noise_levels_.gpu_data(), top_nl);
#ifdef IMAGE_DEBUG
			const Dtype scale = layer_param_.add_noise_param().scale();
			const int h = bottom[i]->height();
			const int w = bottom[i]->width();
			const int c = bottom[i]->channels();
			cv::Mat img;
			img.create(h, w*2, CV_MAKETYPE(CV_8U, c));
			for(int in=0; in<n; in++)
			{
				const Dtype* clean = bottom[i]->cpu_data() + in*np;
				const Dtype* noised = (*top)[i]->cpu_data() + in*np;
				Dtype sum = 0.f, sum2 = 0.f;
				for(int j=0; j<np; j++)
				{
					Dtype ns = noised[j]-clean[j];
					sum += ns;
					sum2 += ns*ns;

					int ci = j/(h*w);
					int y = j%(h*w)/w;
					int x = j%(h*w)%w;
					img.at<unsigned char>(y,x) = std::min(Dtype(255), std::max(Dtype(0), clean[j]/scale));
					img.at<unsigned char>(y,x+w) = std::min(Dtype(255), std::max(Dtype(0), noised[j]/scale));
				}
				Dtype cov = sum2/np - (sum/np)*(sum/np);
				Dtype std = sqrt(cov);
				LOG(INFO) << "nl: " << noise_levels_.cpu_data()[in]/scale << ", " << std/scale;
				cv::imshow("noised", img);
				cvWaitKey();
			}
#endif
		}
		return Dtype(0.);
	}

	INSTANTIATE_CLASS(AddNoiseLayer);

}  // namespace caffe

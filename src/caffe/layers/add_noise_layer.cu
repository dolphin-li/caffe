#pragma once

#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

	template <typename Dtype>
	Dtype AddNoiseLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		vector<Blob<Dtype>*>* top) 
	{
		for(size_t i = 0; i<bottom.size(); i++)
		{
			// generate noise-levels
			generate_noise_levels();
			const Dtype* noise_level_data = noise_levels_.gpu_data();
			const Dtype* bottom_data = bottom[i]->gpu_data();
			Dtype* top_data = (*top)[i]->mutable_gpu_data();
			Dtype* noise_data = noises_.mutable_gpu_data();

			const int n = bottom[i]->num();
			const int np = bottom[i]->channels()*bottom[i]->height()*bottom[i]->width();
			caffe_gpu_rng_gaussian(n*np, Dtype(0), Dtype(1), noise_data);
			caffe_gpu_dgmm(HIPBLAS_SIDE_RIGHT, np, n, noise_data, noise_level_data, noise_data);
			caffe_gpu_add(n*np, bottom_data, noise_data, top_data);
		}
		return Dtype(0.);
	}

	INSTANTIATE_CLASS(AddNoiseLayer);

}  // namespace caffe

#include "hip/hip_runtime.h"
#pragma once

#include <hip/hip_runtime.h>

#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"

//#define IMAGE_DEBUG
#ifdef IMAGE_DEBUG
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#endif

namespace caffe {

	template <typename Dtype>
	__global__ void rand_sample_patches_gpu(const Dtype* src, Dtype* dst, const unsigned int* randXY, int nimg, int nsample, int c, int h, int w, int sz)
	{
		const int sz2 = sz*sz;
		const int pixel_per_img_sample = nsample*sz2;
		CUDA_KERNEL_LOOP(index, nimg*pixel_per_img_sample) {
			const int iimg = index / pixel_per_img_sample;
			const int idx_per_img = index - iimg*pixel_per_img_sample;
			const int isample = idx_per_img / sz2;
			const int ipixel = idx_per_img - sz2 * isample;
			const int iy = ipixel / sz;
			const int ix = ipixel - sz * iy;
			const int rndpos = iimg * nsample + isample;
			unsigned int x0 = randXY[rndpos*2] % (w-sz);
			unsigned int y0 = randXY[rndpos*2+1] % (h-sz);

			const Dtype* pSrc = src + iimg*c*h*w + (y0+iy)*w + x0 + ix;
			Dtype* pDst = dst + iimg*pixel_per_img_sample  + isample*sz2*c + iy*sz + ix;
			for(int ci=0; ci<c; ci++)
				pDst[ci*sz2] = pSrc[ci*h*w];
		}
	}

	template <typename Dtype>
	Dtype PatchSampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		vector<Blob<Dtype>*>* top) 
	{
		const int n_per_img = layer_param_.patch_sample_param().sample_per_img();
		const int sz = layer_param_.patch_sample_param().patch_size();
		for(size_t i = 0; i<bottom.size(); i++)
		{
			const Dtype* bottom_data = bottom[i]->gpu_data();
			Dtype* top_data = (*top)[i]->mutable_gpu_data();
			const int n = bottom[i]->num();
			const int c = bottom[i]->channels();
			const int h = bottom[i]->height();
			const int w = bottom[i]->width();
			unsigned int* rand_xy_buffer = 0;
			CUDA_CHECK( hipMalloc<unsigned int>(&rand_xy_buffer, n*n_per_img*2*sizeof(unsigned int)) );

			CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), rand_xy_buffer, n*n_per_img*2));
			rand_sample_patches_gpu<Dtype><<<CAFFE_GET_BLOCKS(n*n_per_img*sz*sz), CAFFE_CUDA_NUM_THREADS>>>
				(bottom_data, top_data, rand_xy_buffer, n, n_per_img, c, h, w, sz);
			CUDA_POST_KERNEL_CHECK;

#ifdef IMAGE_DEBUG
			const int cl = sz*sz;
			cv::Mat img;
			img.create(h, w, CV_8UC1);
			std::vector<unsigned int> rand_xy_cpu(n*n_per_img*2);
			CUDA_CHECK( hipMemcpy(rand_xy_cpu.data(), rand_xy_buffer, n*n_per_img*2*sizeof(unsigned int), hipMemcpyDeviceToHost) );
			for(int ni=0; ni<(*top)[i]->num(); ni++)
			{
				const Dtype* pSrc = (*top)[i]->cpu_data() + ni*sz*sz*c;
				for(int p=0; p<cl; p++)
				{
					unsigned int x0 = rand_xy_cpu[ni*2] % (w-sz);
					unsigned int y0 = rand_xy_cpu[ni*2+1] % (h-sz);
					img.at<char>(p/sz + y0, p%sz + x0) = pSrc[p]*255;
				}
				if(ni%n_per_img==n_per_img-1)
				{
					cv::imshow("patch", img);
					memset(img.data, 0, img.dataend-img.data);
					cvWaitKey();
				}
			}
#endif

			CUDA_CHECK( hipFree(rand_xy_buffer) );
		}
		return Dtype(0.);
	}

	INSTANTIATE_CLASS(PatchSampleLayer);

}  // namespace caffe

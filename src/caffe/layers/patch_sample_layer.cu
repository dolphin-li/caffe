#include "hip/hip_runtime.h"
#pragma once

#include "caffe/vision_layers.hpp"
#include "caffe/util/math_functions.hpp"
namespace caffe {

	template <typename Dtype>
	__global__ void rand_sample_patches_gpu(const Dtype* src, Dtype* dst, const unsigned int* randXY, int nsample, int c, int h, int w, int sz)
	{
		const int sz2 = sz*sz;
		CUDA_KERNEL_LOOP(index, nsample*sz2) {
			int isample = index / sz2;
			int ipixel = index % sz2;
			int ix = ipixel % sz;
			int iy = ipixel / sz;
			unsigned int x0 = randXY[isample*2] % (w-sz);
			unsigned int y0 = randXY[isample*2+1] % (h-sz);
			for(int ci=0; ci<c; ci++)
			{
				const Dtype* pSrc = src + ci*h*w + (y0+iy)*w + x0 + ix;
				Dtype* pDst = dst + ci*sz*sz + iy*sz + ix;
				*pDst = *pSrc;
			}
		}
	}

	template <typename Dtype>
	Dtype PatchSampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
		vector<Blob<Dtype>*>* top) 
	{
		const int n_per_img = layer_param_.patch_sample_param().sample_per_img();
		const int sz = layer_param_.patch_sample_param().patch_size();
		unsigned int* rand_xy_buffer = 0;
		CUDA_CHECK( hipMalloc<unsigned int>(&rand_xy_buffer, n_per_img*2) );
		shared_ptr<unsigned int> rand_xy(rand_xy_buffer);
		for(size_t i = 0; i<bottom.size(); i++)
		{
			const Dtype* bottom_data = bottom[i]->gpu_data();
			Dtype* top_data = (*top)[i]->mutable_gpu_data();
			const int n = bottom[i]->num();
			const int c = bottom[i]->channels();
			const int h = bottom[i]->height();
			const int w = bottom[i]->width();
			for(int ni=0; ni<n; ni++)
			{
				Dtype* pDst = top_data + ni*n_per_img*c*sz*sz;
				const Dtype* pSrc = bottom_data + ni*c*h*w;
				CURAND_CHECK(hiprandGenerate(Caffe::curand_generator(), rand_xy.get(), n_per_img*2));
				rand_sample_patches_gpu<Dtype><<<CAFFE_GET_BLOCKS(n_per_img*sz*sz), CAFFE_CUDA_NUM_THREADS>>>
					(pSrc, pDst, rand_xy.get(), n_per_img, c, h, w, sz);
				CUDA_POST_KERNEL_CHECK;
			}
		}
		return Dtype(0.);
	}

	INSTANTIATE_CLASS(PatchSampleLayer);

}  // namespace caffe
